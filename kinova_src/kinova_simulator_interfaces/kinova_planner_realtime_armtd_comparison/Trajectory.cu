#include "hip/hip_runtime.h"
#ifndef TRAJECTORY_CPP
#define TRAJECTORY_CPP

#include "Trajectory.h"

ConstantAccelerationCurve::ConstantAccelerationCurve(double* q0_inp, double* qd0_inp, 
                                                     double* c_cos_q_des_inp, double* g_cos_q_des_inp, double* r_cos_q_des_inp,
                                                     double* c_sin_q_des_inp, double* g_sin_q_des_inp, double* r_sin_q_des_inp,
                                                     double* k_range_inp) {
    q0 = q0_inp;
    qd0 = qd0_inp; 

    c_cos_q_des = c_cos_q_des_inp;
    g_cos_q_des = g_cos_q_des_inp;
    r_cos_q_des = r_cos_q_des_inp;

    c_sin_q_des = c_sin_q_des_inp;
    g_sin_q_des = g_sin_q_des_inp;
    r_sin_q_des = r_sin_q_des_inp;

    R = PZsparseArray(NUM_JOINTS + 1, NUM_TIME_STEPS);
    R_t = PZsparseArray(NUM_JOINTS, NUM_TIME_STEPS);

    k_range = k_range_inp;

    dt = 1.0 / NUM_TIME_STEPS;
}

void ConstantAccelerationCurve::makePolyZono(int t_ind) {
    assert(t_ind < NUM_TIME_STEPS);

    for (int i = 0; i < NUM_FACTORS; i++) {
        const double k_range_elt = k_range[i];
        const double cos_q0 = cos(q0[i]);
        const double sin_q0 = sin(q0[i]);

        // cos(q_des)
        double cos_q_des_center = cos_q0 * c_cos_q_des[i * NUM_TIME_STEPS + t_ind] - sin_q0 * c_sin_q_des[i * NUM_TIME_STEPS + t_ind];
        double cos_q_des_coeff[2];
        cos_q_des_coeff[0] = cos_q0 * g_cos_q_des[i * NUM_TIME_STEPS + t_ind] - sin_q0 * g_sin_q_des[i * NUM_TIME_STEPS + t_ind];
        cos_q_des_coeff[1] = fabs(cos_q0) * r_cos_q_des[i * NUM_TIME_STEPS + t_ind] + fabs(sin_q0) * r_sin_q_des[i * NUM_TIME_STEPS + t_ind];

        cos_q_des_coeff[1] *= 4.0;

        uint64_t cos_q_des_degree[2][NUM_FACTORS * 6] = {0};
        cos_q_des_degree[0][i] = 1; // k
        cos_q_des_degree[1][i + NUM_FACTORS * 4] = 1; // cosqe
        // cos_q_des[t_ind * NUM_FACTORS + i] = PZsparse(cos_q_des_center, cos_q_des_coeff, cos_q_des_degree, 2);

        // sin(q_des)
        double sin_q_des_center = cos_q0 * c_sin_q_des[i * NUM_TIME_STEPS + t_ind] + sin_q0 * c_cos_q_des[i * NUM_TIME_STEPS + t_ind];
        double sin_q_des_coeff[2];
        sin_q_des_coeff[0] = cos_q0 * g_sin_q_des[i * NUM_TIME_STEPS + t_ind] + sin_q0 * g_cos_q_des[i * NUM_TIME_STEPS + t_ind];
        sin_q_des_coeff[1] = fabs(cos_q0) * r_sin_q_des[i * NUM_TIME_STEPS + t_ind] + fabs(sin_q0) * r_cos_q_des[i * NUM_TIME_STEPS + t_ind];

        sin_q_des_coeff[1] *= 4.0;

        uint64_t sin_q_des_degree[2][NUM_FACTORS * 6] = {0};
        sin_q_des_degree[0][i] = 1; // k
        sin_q_des_degree[1][i + NUM_FACTORS * 5] = 1; // sinqe
        // sin_q_des[t_ind * NUM_FACTORS + i] = PZsparse(sin_q_des_center, sin_q_des_coeff, sin_q_des_degree, 2);

        R(i, t_ind) = PZsparse(rots[i * 3], rots[i * 3 + 1], rots[i * 3 + 2]);

        if (axes[i] != 0) {
            R(i, t_ind) = R(i, t_ind) * PZsparse(cos_q_des_center, cos_q_des_coeff, cos_q_des_degree, 2,
                                                 sin_q_des_center, sin_q_des_coeff, sin_q_des_degree, 2,
                                                 axes[i]);
        }

        R_t(i, t_ind) = R(i, t_ind).transpose();
    }

    // assume all fixed joints are at the end of the kinematics chain
    for (int i = NUM_FACTORS; i < NUM_JOINTS; i++) {
        R(i, t_ind) = PZsparse(rots[i * 3], rots[i * 3 + 1], rots[i * 3 + 2]);
        R_t(i, t_ind) = R(i, t_ind).transpose();
    }

    R(NUM_JOINTS, t_ind) = PZsparse(0, 0, 0);
}

void ConstantAccelerationCurve::returnJointStateExtremum(double* extremum, const double* k) const {
    double t_move = 0.5;
    double t_total = 1.0;
    double t_to_stop = t_total - t_move;

    for (int i = 0; i < NUM_FACTORS; i++){
        double k_actual = k_range[i] * k[i];
        double q_peak = q0[i] + qd0[i] * t_move + k_actual * t_move * t_move * 0.5;
        double q_dot_peak = qd0[i] + k_actual * t_move;
        double q_ddot_to_stop = -q_dot_peak / t_to_stop;
        double q_stop = q_peak + q_dot_peak * t_to_stop + 0.5 * q_ddot_to_stop * t_to_stop * t_to_stop;
        double t_max_min_to_peak = -qd0[i] / k_actual;

        double q_max_to_peak;
        double q_min_to_peak;
        double q_dot_max_to_peak;
        double q_dot_min_to_peak;

        double grad_q_max_to_peak;
        double grad_q_min_to_peak;
        double grad_q_dot_max_to_peak;
        double grad_q_dot_min_to_peak;

        double q_max_to_stop;
        double q_min_to_stop;
        double q_dot_max_to_stop;
        double q_dot_min_to_stop;

        double grad_q_max_to_stop;
        double grad_q_min_to_stop;
        double grad_q_dot_max_to_stop;
        double grad_q_dot_min_to_stop;

        double q_endpoints_ordered[2];
        double grad_q_endpoints_ordered[2];

        if (q_peak >= q0[i]){
            q_endpoints_ordered[0] = q0[i]; 
            q_endpoints_ordered[1] = q_peak;
            grad_q_endpoints_ordered[0] = 0; 
            grad_q_endpoints_ordered[1] = 0.5 * t_move * t_move;
        }
        else{
            q_endpoints_ordered[0] = q_peak; 
            q_endpoints_ordered[1] = q0[i];
            grad_q_endpoints_ordered[0] = 0.5 * t_move * t_move; 
            grad_q_endpoints_ordered[1] = 0;
        }

        if (t_max_min_to_peak > 0 && t_max_min_to_peak < t_move){
            if (k_actual >= 0){
                q_min_to_peak = q0[i] + qd0[i] * t_max_min_to_peak + 0.5 * k_actual * t_max_min_to_peak * t_max_min_to_peak;
                q_max_to_peak  = q_endpoints_ordered[1];
                grad_q_min_to_peak = (0.5 * qd0[i] * qd0[i]) / (k_actual * k_actual);
                grad_q_max_to_peak = grad_q_endpoints_ordered[1];
            }
            else{
                q_min_to_peak = q_endpoints_ordered[0];
                q_max_to_peak = q0[i] + qd0[i] * t_max_min_to_peak + 0.5 * k_actual * t_max_min_to_peak * t_max_min_to_peak;
                grad_q_min_to_peak = grad_q_endpoints_ordered[0];
                grad_q_max_to_peak = (0.5 * qd0[i] * qd0[i]) / (k_actual * k_actual);
            }
        }
        else{
            q_min_to_peak = q_endpoints_ordered[0];
            q_max_to_peak = q_endpoints_ordered[1];
            
            grad_q_min_to_peak = grad_q_endpoints_ordered[0];
            grad_q_max_to_peak = grad_q_endpoints_ordered[1];
        }

        if( q_dot_peak >= qd0[i]){
            q_dot_min_to_peak = qd0[i];
            q_dot_max_to_peak = q_dot_peak;
            
            grad_q_dot_min_to_peak = 0;
            grad_q_dot_max_to_peak = t_move;
        }
        else{
            q_dot_min_to_peak = q_dot_peak;
            q_dot_max_to_peak = qd0[i];
            
            grad_q_dot_min_to_peak = t_move;
            grad_q_dot_max_to_peak = 0;
        }

        if( q_stop >= q_peak){
            q_min_to_stop = q_peak;
            q_max_to_stop = q_stop;
            
            grad_q_min_to_stop = 0.5 * t_move * t_move;
            grad_q_max_to_stop = 0.5 * t_move * t_move + 0.5 * t_move * t_to_stop;
        }
        else{
            q_min_to_stop = q_stop;
            q_max_to_stop = q_peak;
            
            grad_q_min_to_stop = 0.5 * t_move * t_move + 0.5 * t_move * t_to_stop;
            grad_q_max_to_stop = 0.5 * t_move * t_move;
        }

        if(q_dot_peak >= 0){
            q_dot_min_to_stop = 0;
            q_dot_max_to_stop = q_dot_peak;
            
            grad_q_dot_min_to_stop = 0;
            grad_q_dot_max_to_stop = t_move;
        }
        else{
            q_dot_min_to_stop = q_dot_peak;
            q_dot_max_to_stop = 0;
            
            grad_q_dot_min_to_stop = t_move;
            grad_q_dot_max_to_stop = 0;
        }

        if (q_min_to_peak <= q_min_to_stop){
            extremum[i] = q_min_to_peak; // q_min[i]
        }
        else{
            extremum[i] = q_min_to_stop; // q_min[i]
        }

        if (q_max_to_peak >= q_max_to_stop){
            extremum[i + NUM_FACTORS] = q_max_to_peak; // q_max[i]
        }
        else{
            extremum[i + NUM_FACTORS] = q_max_to_stop; // q_max[i]
        }

        if (q_dot_min_to_peak <= q_dot_min_to_stop){
            extremum[i + 2 * NUM_FACTORS] = q_dot_min_to_peak; // q_dot_min[i]
        }
        else{
            extremum[i + 2 * NUM_FACTORS] = q_dot_min_to_stop; // q_dot_min[i]
        }

        if (q_dot_max_to_peak >= q_dot_max_to_stop){
            extremum[i + 3 * NUM_FACTORS] = q_dot_max_to_peak; // q_dot_max[i]
        }
        else{
            extremum[i + 3 * NUM_FACTORS] = q_dot_max_to_stop; // q_dot_max[i]
        }
    }
}

void ConstantAccelerationCurve::returnJointStateExtremumGradient(double* extremumGradient, const double* k) const {
    memset(extremumGradient, 0, 4 * NUM_FACTORS * NUM_FACTORS);

    double t_move = 0.5;
    double t_total = 1.0;
    double t_to_stop = t_total - t_move;

    for (int i = 0; i < NUM_FACTORS; i++){
        double k_actual = k_range[i] * k[i];
        double q_peak = q0[i] + qd0[i] * t_move + k_actual * t_move * t_move * 0.5;
        double q_dot_peak = qd0[i] + k_actual * t_move;
        double q_ddot_to_stop = -q_dot_peak / t_to_stop;
        double q_stop = q_peak + q_dot_peak * t_to_stop + 0.5 * q_ddot_to_stop * t_to_stop * t_to_stop;
        double t_max_min_to_peak = -qd0[i] / k_actual;

        double q_max_to_peak;
        double q_min_to_peak;
        double q_dot_max_to_peak;
        double q_dot_min_to_peak;

        double grad_q_max_to_peak;
        double grad_q_min_to_peak;
        double grad_q_dot_max_to_peak;
        double grad_q_dot_min_to_peak;

        double q_max_to_stop;
        double q_min_to_stop;
        double q_dot_max_to_stop;
        double q_dot_min_to_stop;

        double grad_q_max_to_stop;
        double grad_q_min_to_stop;
        double grad_q_dot_max_to_stop;
        double grad_q_dot_min_to_stop;

        double q_endpoints_ordered[2];
        double grad_q_endpoints_ordered[2];

        if (q_peak >= q0[i]){
            q_endpoints_ordered[0] = q0[i]; 
            q_endpoints_ordered[1] = q_peak;
            grad_q_endpoints_ordered[0] = 0; 
            grad_q_endpoints_ordered[1] = 0.5 * t_move * t_move;
        }
        else{
            q_endpoints_ordered[0] = q_peak; 
            q_endpoints_ordered[1] = q0[i];
            grad_q_endpoints_ordered[0] = 0.5 * t_move * t_move; 
            grad_q_endpoints_ordered[1] = 0;
        }

        if (t_max_min_to_peak > 0 && t_max_min_to_peak < t_move){
            if (k_actual >= 0){
                q_min_to_peak = q0[i] + qd0[i] * t_max_min_to_peak + 0.5 * k_actual * t_max_min_to_peak * t_max_min_to_peak;
                q_max_to_peak  = q_endpoints_ordered[1];
                grad_q_min_to_peak = (0.5 * qd0[i] * qd0[i]) / (k_actual * k_actual);
                grad_q_max_to_peak = grad_q_endpoints_ordered[1];
            }
            else{
                q_min_to_peak = q_endpoints_ordered[0];
                q_max_to_peak = q0[i] + qd0[i] * t_max_min_to_peak + 0.5 * k_actual * t_max_min_to_peak * t_max_min_to_peak;
                grad_q_min_to_peak = grad_q_endpoints_ordered[0];
                grad_q_max_to_peak = (0.5 * qd0[i] * qd0[i]) / (k_actual * k_actual);
            }
        }
        else{
            q_min_to_peak = q_endpoints_ordered[0];
            q_max_to_peak = q_endpoints_ordered[1];
            
            grad_q_min_to_peak = grad_q_endpoints_ordered[0];
            grad_q_max_to_peak = grad_q_endpoints_ordered[1];
        }

        if( q_dot_peak >= qd0[i]){
            q_dot_min_to_peak = qd0[i];
            q_dot_max_to_peak = q_dot_peak;
            
            grad_q_dot_min_to_peak = 0;
            grad_q_dot_max_to_peak = t_move;
        }
        else{
            q_dot_min_to_peak = q_dot_peak;
            q_dot_max_to_peak = qd0[i];
            
            grad_q_dot_min_to_peak = t_move;
            grad_q_dot_max_to_peak = 0;
        }

        if( q_stop >= q_peak){
            q_min_to_stop = q_peak;
            q_max_to_stop = q_stop;
            
            grad_q_min_to_stop = 0.5 * t_move * t_move;
            grad_q_max_to_stop = 0.5 * t_move * t_move + 0.5 * t_move * t_to_stop;
        }
        else{
            q_min_to_stop = q_stop;
            q_max_to_stop = q_peak;
            
            grad_q_min_to_stop = 0.5 * t_move * t_move + 0.5 * t_move * t_to_stop;
            grad_q_max_to_stop = 0.5 * t_move * t_move;
        }

        if(q_dot_peak >= 0){
            q_dot_min_to_stop = 0;
            q_dot_max_to_stop = q_dot_peak;
            
            grad_q_dot_min_to_stop = 0;
            grad_q_dot_max_to_stop = t_move;
        }
        else{
            q_dot_min_to_stop = q_dot_peak;
            q_dot_max_to_stop = 0;
            
            grad_q_dot_min_to_stop = t_move;
            grad_q_dot_max_to_stop = 0;
        }

        if (q_min_to_peak <= q_min_to_stop){
            // extremum[i] = q_min_to_peak; // q_min[i]
            extremumGradient[i * NUM_FACTORS + i] = grad_q_min_to_peak;
        }
        else{
            // extremum[i] = q_min_to_stop; // q_min[i]
            extremumGradient[i * NUM_FACTORS + i] = grad_q_min_to_stop;
        }

        if (q_max_to_peak >= q_max_to_stop){
            // extremum[i + NUM_FACTORS] = q_max_to_peak; // q_max[i]
            extremumGradient[(i + NUM_FACTORS) * NUM_FACTORS + i] = grad_q_max_to_peak;
        }
        else{
            // extremum[i + NUM_FACTORS] = q_max_to_stop; // q_max[i]
            extremumGradient[(i + NUM_FACTORS) * NUM_FACTORS + i] = grad_q_max_to_stop;
        }

        if (q_dot_min_to_peak <= q_dot_min_to_stop){
            // extremum[i + 2 * NUM_FACTORS] = q_dot_min_to_peak; // q_dot_min[i]
            extremumGradient[(i + 2 * NUM_FACTORS) * NUM_FACTORS + i] = grad_q_dot_min_to_peak;
        }
        else{
            // extremum[i + 2 * NUM_FACTORS] = q_dot_min_to_stop; // q_dot_min[i]
            extremumGradient[(i + 2 * NUM_FACTORS) * NUM_FACTORS + i] = grad_q_dot_min_to_stop;
        }

        if (q_dot_max_to_peak >= q_dot_max_to_stop){
            // extremum[i + 3 * NUM_FACTORS] = q_dot_max_to_peak; // q_dot_max[i]
            extremumGradient[(i + 3 * NUM_FACTORS) * NUM_FACTORS + i] = grad_q_dot_max_to_peak;
        }
        else{
            // extremum[i + 3 * NUM_FACTORS] = q_dot_max_to_stop; // q_dot_max[i]
            extremumGradient[(i + 3 * NUM_FACTORS) * NUM_FACTORS + i] = grad_q_dot_max_to_stop;
        }
    }
}

#endif