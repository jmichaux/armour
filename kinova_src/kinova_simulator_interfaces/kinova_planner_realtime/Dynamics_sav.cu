#ifndef DYNAMICS_CPP
#define DYNAMICS_CPP

#include "Dynamics.h"

KinematicsDynamics::KinematicsDynamics(BezierCurve* traj_input) {
    traj = traj_input;

    // pre-allocate memory
    links = PZsparseArray(NUM_FACTORS * 3, NUM_TIME_STEPS);
    mass_nominal_arr = PZsparseArray(NUM_JOINTS, 1);
    mass_uncertain_arr = PZsparseArray(NUM_JOINTS, 1);
    I_nominal_arr = PZsparseArray(NUM_JOINTS, 1);
    I_uncertain_arr = PZsparseArray(NUM_JOINTS, 1);
    u_nom = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);
    u_nom_int = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);
    f_c_int = PZsparseArray(1,NUM_TIME_STEPS);
    n_c_int = PZsparseArray(1,NUM_TIME_STEPS);
    f_c_nom = PZsparseArray(1,NUM_TIME_STEPS);
    n_c_nom = PZsparseArray(1,NUM_TIME_STEPS);
    r = PZsparseArray(NUM_FACTORS, 1);
    Mr = PZsparseArray(NUM_FACTORS, NUM_TIME_STEPS);

    // initialize robot properties
    for (int i = 0; i < NUM_JOINTS; i++) {
        trans_matrix(i, 0) = Eigen::MatrixXd::Zero(3, 1);
        trans_matrix(i, 0)(0) = trans[3 * i];
        trans_matrix(i, 0)(1) = trans[3 * i + 1];
        trans_matrix(i, 0)(2) = trans[3 * i + 2];

        // com_matrix(i, 0) = Eigen::MatrixXd::Zero(3, 1);
        // com_matrix(i, 0)(0) = com[i][0];
        // com_matrix(i, 0)(1) = com[i][1];
        // com_matrix(i, 0)(2) = com[i][2];

        Eigen::MatrixXd mass_matrix(1, 1);
        mass_matrix(0) = mass[i];
        mass_nominal_arr(i) = PZsparse(mass_matrix);
        mass_uncertain_arr(i) = PZsparse(mass_matrix, mass_uncertainty[i]);

        Eigen::Matrix3d inertia_matrix;
        for (int j = 0; j < 9; j++) {
            inertia_matrix(j) = inertia[i * 9 + j]; // This may not be right...
        }
        I_nominal_arr(i) = PZsparse(inertia_matrix);
        I_uncertain_arr(i) = PZsparse(inertia_matrix, inertia_uncertainty[i]);

        if (i < NUM_FACTORS) {
            r(i) = PZsparse(0, Interval(-eps, eps));
        }
    }

    trans_matrix(NUM_JOINTS, 0) = Eigen::MatrixXd::Zero(3, 1);
    trans_matrix(NUM_JOINTS, 0)(0) = trans[3 * NUM_JOINTS];
    trans_matrix(NUM_JOINTS, 0)(1) = trans[3 * NUM_JOINTS + 1];
    trans_matrix(NUM_JOINTS, 0)(2) = trans[3 * NUM_JOINTS + 2];

    // define original link PZs
    links = PZsparseArray(NUM_JOINTS, NUM_TIME_STEPS);

    for (int i = 0; i < NUM_JOINTS; i++) {
        PZsparseArray link(3, 1);

        for (int j = 0; j < 3; j++) {
            uint64_t degree[1][NUM_FACTORS * 6] = {0};
            degree[0][NUM_FACTORS * (j + 1)] = 1; // use qde, qdae, qdde for x, y, z generator
            double temp = link_zonotope_generators[i][j];
            link(j, 0) = PZsparse(link_zonotope_center[i][j], &temp, degree, 1);
        }

        links(i, 0) = stack(link);

        for (int j = 1; j < NUM_TIME_STEPS; j++) {
            links(i, j) = links(i, 0);
        }
    }
}

void KinematicsDynamics::fk(uint s_ind) {
    PZsparse FK_R = PZsparse(0, 0, 0); // identity matrix
    PZsparse FK_T(3, 1);
    int j = 0;

    for (int i = 0; i < NUM_JOINTS; i++) {
        PZsparse P(trans_matrix(i, 0));
        
        FK_T = FK_T + FK_R * P;
        FK_R = FK_R * traj->R(i, s_ind);
        
        links(i, s_ind) = FK_R * links(i, s_ind) + FK_T;
    }
}

void KinematicsDynamics::rnea(uint s_ind,
                              PZsparseArray& mass_arr,
                              PZsparseArray& I_arr,
                              PZsparseArray& u,
                              PZsparseArray& f_c,
                              PZsparseArray& n_c,
                              bool setGravity) {
    PZsparse& cq1 = traj->cos_q_des(0, s_ind);
    PZsparse& cq2 = traj->cos_q_des(1, s_ind);
    PZsparse& cq3 = traj->cos_q_des(2, s_ind);
    PZsparse& cq4 = traj->cos_q_des(3, s_ind);
    PZsparse& cq5 = traj->cos_q_des(4, s_ind);
    PZsparse& cq6 = traj->cos_q_des(5, s_ind);
    PZsparse& cq7 = traj->cos_q_des(6, s_ind);

    PZsparse& sq1 = traj->sin_q_des(0, s_ind);
    PZsparse& sq2 = traj->sin_q_des(1, s_ind);
    PZsparse& sq3 = traj->sin_q_des(2, s_ind);
    PZsparse& sq4 = traj->sin_q_des(3, s_ind);
    PZsparse& sq5 = traj->sin_q_des(4, s_ind);
    PZsparse& sq6 = traj->sin_q_des(5, s_ind);
    PZsparse& sq7 = traj->sin_q_des(6, s_ind);

    PZsparse& qd1 = traj->qd_des(0, s_ind);
    PZsparse& qd2 = traj->qd_des(1, s_ind);
    PZsparse& qd3 = traj->qd_des(2, s_ind);
    PZsparse& qd4 = traj->qd_des(3, s_ind);
    PZsparse& qd5 = traj->qd_des(4, s_ind);
    PZsparse& qd6 = traj->qd_des(5, s_ind);
    PZsparse& qd7 = traj->qd_des(6, s_ind);

    PZsparse& qda1 = traj->qda_des(0, s_ind);
    PZsparse& qda2 = traj->qda_des(1, s_ind);
    PZsparse& qda3 = traj->qda_des(2, s_ind);
    PZsparse& qda4 = traj->qda_des(3, s_ind);
    PZsparse& qda5 = traj->qda_des(4, s_ind);
    PZsparse& qda6 = traj->qda_des(5, s_ind);
    PZsparse& qda7 = traj->qda_des(6, s_ind);

    PZsparse& qdd1 = traj->qdda_des(0, s_ind);
    PZsparse& qdd2 = traj->qdda_des(1, s_ind);
    PZsparse& qdd3 = traj->qdda_des(2, s_ind);
    PZsparse& qdd4 = traj->qdda_des(3, s_ind);
    PZsparse& qdd5 = traj->qdda_des(4, s_ind);
    PZsparse& qdd6 = traj->qdda_des(5, s_ind);
    PZsparse& qdd7 = traj->qdda_des(6, s_ind);

    PZsparse w1(0);
    PZsparse w2(0);
    PZsparse w3(0);
    PZsparse w_aux1(0);
    PZsparse w_aux2(0);
    PZsparse w_aux3(0);
    PZsparse wdot1(0);
    PZsparse wdot2(0);
    PZsparse wdot3(0);
    PZsparse linear_acc1(0);
    PZsparse linear_acc2(0);
    PZsparse linear_acc3(0);

    PZsparse w_new1(0);
    PZsparse w_new2(0);
    PZsparse w_new3(0);
    PZsparse w_aux_new1(0);
    PZsparse w_aux_new2(0);
    PZsparse w_aux_new3(0);
    PZsparse wdot_new1(0);
    PZsparse wdot_new2(0);
    PZsparse wdot_new3(0);
    PZsparse linear_acc_new1(0);
    PZsparse linear_acc_new2(0);
    PZsparse linear_acc_new3(0);

    PZsparse t2(0);
    PZsparse t3(0);
    PZsparse t4(0);
    PZsparse t5(0);
    PZsparse t6(0);
    PZsparse t7(0);
    PZsparse t8(0);
    PZsparse t9(0);
    PZsparse t10(0);
    PZsparse t11(0);
    PZsparse t12(0);
    PZsparse t13(0);
    PZsparse t14(0);
    PZsparse t15(0);
    PZsparse t16(0);
    PZsparse t17(0);
    PZsparse t18(0);
    PZsparse t19(0);
    PZsparse t20(0);
    PZsparse t21(0);
    PZsparse t22(0);

    // joint 1
    w_new3 = qd1;

    w_aux_new3 = qda1;

    wdot_new3 = qdd1;

    linear_acc_new3 = -9.81E+2/1.0E+2;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[0][0]*w_aux2;
    t3 = com[0][0]*w_aux3;
    t4 = com[0][1]*w_aux1;
    t5 = com[0][1]*w_aux3;
    t6 = com[0][2]*w_aux1;
    t7 = com[0][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F1_1 = -mass_arr(0,0)*(-linear_acc1+com[0][1]*wdot3-com[0][2]*wdot2+t11*w2+t12*w3);
    PZsparse F1_2 = mass_arr(0,0)*(linear_acc2+com[0][0]*wdot3-com[0][2]*wdot1+t11*w1-t13*w3);
    PZsparse F1_3 = mass_arr(0,0)*(linear_acc3-com[0][0]*wdot2+com[0][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(0,0)(0,0)*w1;
    t3 = I_arr(0,0)(0,1)*w2;
    t4 = I_arr(0,0)(0,2)*w3;
    t5 = I_arr(0,0)(1,0)*w1;
    t6 = I_arr(0,0)(1,1)*w2;
    t7 = I_arr(0,0)(1,2)*w3;
    t8 = I_arr(0,0)(2,0)*w1;
    t9 = I_arr(0,0)(2,1)*w2;
    t10 = I_arr(0,0)(2,2)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N1_1 = I_arr(0,0)(0,0)*wdot1+I_arr(0,0)(0,1)*wdot2+I_arr(0,0)(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N1_2 = I_arr(0,0)(1,0)*wdot1+I_arr(0,0)(1,1)*wdot2+I_arr(0,0)(1,2)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N1_3 = I_arr(0,0)(2,0)*wdot1+I_arr(0,0)(2,1)*wdot2+I_arr(0,0)(2,2)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 2
    w_new1 = cq2*w1-sq2*w2*3.673205103346574E-6+sq2*w3;
    w_new2 = cq2*w2*(-3.673205103346574E-6)+cq2*w3-sq2*w1;
    w_new3 = qd2-w2-w3*3.673205103346574E-6;

    w_aux_new1 = cq2*w_aux1-sq2*w_aux2*3.673205103346574E-6+sq2*w_aux3;
    w_aux_new2 = cq2*w_aux2*(-3.673205103346574E-6)+cq2*w_aux3-sq2*w_aux1;
    w_aux_new3 = qda2-w_aux2-w_aux3*3.673205103346574E-6;

    wdot_new1 = cq2*wdot1-sq2*wdot2*3.673205103346574E-6+sq2*wdot3-qd2*(cq2*w_aux2*3.673205103346574E-6-cq2*w_aux3+sq2*w_aux1);
    wdot_new2 = cq2*wdot2*(-3.673205103346574E-6)+cq2*wdot3-sq2*wdot1-qd2*(cq2*w_aux1-sq2*w_aux2*3.673205103346574E-6+sq2*w_aux3);
    wdot_new3 = qdd2-wdot2-wdot3*3.673205103346574E-6;

    t2 = -linear_acc1;
    t3 = w_aux3*5.375E-3;
    t4 = wdot1*5.375E-3;
    t5 = wdot3*5.375E-3;
    t6 = w_aux1*w1*5.375E-3;
    t7 = w_aux1*w2*5.375E-3;
    t10 = w_aux2*1.2838E-1;
    t11 = wdot1*1.2838E-1;
    t12 = wdot2*1.2838E-1;
    t13 = w_aux1*w1*1.2838E-1;
    t14 = w_aux1*w3*1.2838E-1;
    t8 = -t6;
    t9 = -t7;
    t15 = t3+t10;
    t16 = t15*w2;
    t17 = t15*w3;
    t20 = t2+t5+t9+t12+t14;
    t18 = -t17;
    t19 = linear_acc3+t4+t13+t16;
    t21 = linear_acc2+t8+t11+t18;
    linear_acc_new1 = -cq2*t20+sq2*t19-sq2*t21*3.673205103346574E-6;
    linear_acc_new2 = cq2*t19-cq2*t21*3.673205103346574E-6+sq2*t20;
    linear_acc_new3 = -linear_acc2-linear_acc3*3.673205103346574E-6-t16*3.673205103346574E-6+t17-wdot1*1.283800197434774E-1+w_aux1*w1*5.374528433928832E-3;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[1][0]*w_aux2;
    t3 = com[1][0]*w_aux3;
    t4 = com[1][1]*w_aux1;
    t5 = com[1][1]*w_aux3;
    t6 = com[1][2]*w_aux1;
    t7 = com[1][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F2_1 = -mass_arr(1,0)*(-linear_acc1+com[1][1]*wdot3-com[1][2]*wdot2+t11*w2+t12*w3);
    PZsparse F2_2 = mass_arr(1,0)*(linear_acc2+com[1][0]*wdot3-com[1][2]*wdot1+t11*w1-t13*w3);
    PZsparse F2_3 = mass_arr(1,0)*(linear_acc3-com[1][0]*wdot2+com[1][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(1,0)(0,0)*w1;
    t3 = I_arr(1,0)(0,1)*w2;
    t4 = I_arr(1,0)(0,2)*w3;
    t5 = I_arr(1,0)(1,0)*w1;
    t6 = I_arr(1,0)(1,1)*w2;
    t7 = I_arr(1,0)(1,2)*w3;
    t8 = I_arr(1,0)(2,0)*w1;
    t9 = I_arr(1,0)(2,1)*w2;
    t10 = I_arr(1,0)(2,2)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N2_1 = I_arr(1,0)(0,0)*wdot1+I_arr(1,0)(0,1)*wdot2+I_arr(1,0)(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N2_2 = I_arr(1,0)(1,0)*wdot1+I_arr(1,0)(1,1)*wdot2+I_arr(1,0)(1,2)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N2_3 = I_arr(1,0)(2,0)*wdot1+I_arr(1,0)(2,1)*wdot2+I_arr(1,0)(2,2)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 3
    w_new1 = cq3*w1-sq3*w2*3.673205103346574E-6-sq3*w3;
    w_new2 = cq3*w2*(-3.673205103346574E-6)-cq3*w3-sq3*w1;
    w_new3 = qd3+w2-w3*3.673205103346574E-6;

    w_aux_new1 = cq3*w_aux1-sq3*w_aux2*3.673205103346574E-6-sq3*w_aux3;
    w_aux_new2 = cq3*w_aux2*(-3.673205103346574E-6)-cq3*w_aux3-sq3*w_aux1;
    w_aux_new3 = qda3+w_aux2-w_aux3*3.673205103346574E-6;

    wdot_new1 = cq3*wdot1-sq3*wdot2*3.673205103346574E-6-sq3*wdot3-qd3*(cq3*w_aux2*3.673205103346574E-6+cq3*w_aux3+sq3*w_aux1);
    wdot_new2 = cq3*wdot2*(-3.673205103346574E-6)-cq3*wdot3-sq3*wdot1+qd3*(-cq3*w_aux1+sq3*w_aux2*3.673205103346574E-6+sq3*w_aux3);
    wdot_new3 = qdd3+wdot2-wdot3*3.673205103346574E-6;

    t2 = -linear_acc1;
    t3 = w_aux2*6.375E-3;
    t4 = wdot1*6.375E-3;
    t5 = wdot2*6.375E-3;
    t6 = w_aux1*w1*6.375E-3;
    t7 = w_aux1*w3*6.375E-3;
    t8 = w_aux3*2.1038E-1;
    t9 = wdot1*2.1038E-1;
    t10 = wdot3*2.1038E-1;
    t11 = w_aux1*w1*2.1038E-1;
    t12 = w_aux1*w2*2.1038E-1;
    t13 = -t8;
    t14 = -t9;
    t15 = -t10;
    t16 = t3+t13;
    t20 = t2+t5+t7+t12+t15;
    t17 = t16*w2;
    t18 = t16*w3;
    t19 = -t18;
    t21 = linear_acc3+t6+t14+t17;
    t22 = linear_acc2+t4+t11+t19;
    linear_acc_new1 = -cq3*t20-sq3*t21-sq3*t22*3.673205103346574E-6;
    linear_acc_new2 = -cq3*t21-cq3*t22*3.673205103346574E-6+sq3*t20;
    linear_acc_new3 = linear_acc2-linear_acc3*3.673205103346574E-6-t17*3.673205103346574E-6+t19+wdot1*6.375772768889642E-3+w_aux1*w1*2.103799765833175E-1;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[2][0]*w_aux2;
    t3 = com[2][0]*w_aux3;
    t4 = com[2][1]*w_aux1;
    t5 = com[2][1]*w_aux3;
    t6 = com[2][2]*w_aux1;
    t7 = com[2][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F3_1 = -mass_arr(2,0)*(-linear_acc1+com[2][1]*wdot3-com[2][2]*wdot2+t11*w2+t12*w3);
    PZsparse F3_2 = mass_arr(2,0)*(linear_acc2+com[2][0]*wdot3-com[2][2]*wdot1+t11*w1-t13*w3);
    PZsparse F3_3 = mass_arr(2,0)*(linear_acc3-com[2][0]*wdot2+com[2][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(2,0)(0,0)*w1;
    t3 = I_arr(2,0)(0,1)*w2;
    t4 = I_arr(2,0)(0,2)*w3;
    t5 = I_arr(2,0)(1,0)*w1;
    t6 = I_arr(2,0)(1,1)*w2;
    t7 = I_arr(2,0)(1,2)*w3;
    t8 = I_arr(2,0)(2,0)*w1;
    t9 = I_arr(2,0)(2,1)*w2;
    t10 = I_arr(2,0)(2,2)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N3_1 = I_arr(2,0)(0,0)*wdot1+I_arr(2,0)(0,1)*wdot2+I_arr(2,0)(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N3_2 = I_arr(2,0)(1,0)*wdot1+I_arr(2,0)(1,1)*wdot2+I_arr(2,0)(1,2)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N3_3 = I_arr(2,0)(2,0)*wdot1+I_arr(2,0)(2,1)*wdot2+I_arr(2,0)(2,2)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 4
    w_new1 = cq4*w1-sq4*w2*3.673205103346573E-6+sq4*w3;
    w_new2 = cq4*w2*(-3.673205103346573E-6)+cq4*w3-sq4*w1;
    w_new3 = qd4-w2-w3*3.673205103346573E-6;

    w_aux_new1 = cq4*w_aux1-sq4*w_aux2*3.673205103346573E-6+sq4*w_aux3;
    w_aux_new2 = cq4*w_aux2*(-3.673205103346573E-6)+cq4*w_aux3-sq4*w_aux1;
    w_aux_new3 = qda4-w_aux2-w_aux3*3.673205103346573E-6;

    wdot_new1 = cq4*wdot1-sq4*wdot2*3.673205103346573E-6+sq4*wdot3-qd4*(cq4*w_aux2*3.673205103346573E-6-cq4*w_aux3+sq4*w_aux1);
    wdot_new2 = cq4*wdot2*(-3.673205103346573E-6)+cq4*wdot3-sq4*wdot1-qd4*(cq4*w_aux1-sq4*w_aux2*3.673205103346573E-6+sq4*w_aux3);
    wdot_new3 = qdd4-wdot2-wdot3*3.673205103346573E-6;

    t2 = -linear_acc1;
    t3 = w_aux3*6.375E-3;
    t4 = wdot1*6.375E-3;
    t5 = wdot3*6.375E-3;
    t6 = w_aux1*w1*6.375E-3;
    t7 = w_aux1*w2*6.375E-3;
    t10 = w_aux2*2.1038E-1;
    t11 = wdot1*2.1038E-1;
    t12 = wdot2*2.1038E-1;
    t13 = w_aux1*w1*2.1038E-1;
    t14 = w_aux1*w3*2.1038E-1;
    t8 = -t6;
    t9 = -t7;
    t15 = t3+t10;
    t16 = t15*w2;
    t17 = t15*w3;
    t20 = t2+t5+t9+t12+t14;
    t18 = -t17;
    t19 = linear_acc3+t4+t13+t16;
    t21 = linear_acc2+t8+t11+t18;
    linear_acc_new1 = -cq4*t20+sq4*t19-sq4*t21*3.673205103346573E-6;
    linear_acc_new2 = cq4*t19-cq4*t21*3.673205103346573E-6+sq4*t20;
    linear_acc_new3 = -linear_acc2-linear_acc3*3.673205103346573E-6-t16*3.673205103346573E-6+t17-wdot1*2.103800234166825E-1+w_aux1*w1*6.374227231110358E-3;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[3][0]*w_aux2;
    t3 = com[3][0]*w_aux3;
    t4 = com[3][1]*w_aux1;
    t5 = com[3][1]*w_aux3;
    t6 = com[3][2]*w_aux1;
    t7 = com[3][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F4_1 = -mass_arr(3,0)*(-linear_acc1+com[3][1]*wdot3-com[3][2]*wdot2+t11*w2+t12*w3);
    PZsparse F4_2 = mass_arr(3,0)*(linear_acc2+com[3][0]*wdot3-com[3][2]*wdot1+t11*w1-t13*w3);
    PZsparse F4_3 = mass_arr(3,0)*(linear_acc3-com[3][0]*wdot2+com[3][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(3,0)(0,0)*w1;
    t3 = I_arr(3,0)(0,1)*w2;
    t4 = I_arr(3,0)(0,2)*w3;
    t5 = I_arr(3,0)(1,0)*w1;
    t6 = I_arr(3,0)(1,1)*w2;
    t7 = I_arr(3,0)(1,2)*w3;
    t8 = I_arr(3,0)(2,0)*w1;
    t9 = I_arr(3,0)(2,1)*w2;
    t10 = I_arr(3,0)(2,2)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N4_1 = I_arr(3,0)(0,0)*wdot1+I_arr(3,0)(0,1)*wdot2+I_arr(3,0)(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N4_2 = I_arr(3,0)(1,0)*wdot1+I_arr(3,0)(1,1)*wdot2+I_arr(3,0)(1,2)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N4_3 = I_arr(3,0)(2,0)*wdot1+I_arr(3,0)(2,1)*wdot2+I_arr(3,0)(2,2)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 5
    w_new1 = cq5*w1-sq5*w2*3.673205103346573E-6-sq5*w3;
    w_new2 = cq5*w2*(-3.673205103346573E-6)-cq5*w3-sq5*w1;
    w_new3 = qd5+w2-w3*3.673205103346573E-6;

    w_aux_new1 = cq5*w_aux1-sq5*w_aux2*3.673205103346573E-6-sq5*w_aux3;
    w_aux_new2 = cq5*w_aux2*(-3.673205103346573E-6)-cq5*w_aux3-sq5*w_aux1;
    w_aux_new3 = qda5+w_aux2-w_aux3*3.673205103346573E-6;

    wdot_new1 = cq5*wdot1-sq5*wdot2*3.673205103346573E-6-sq5*wdot3-qd5*(cq5*w_aux2*3.673205103346573E-6+cq5*w_aux3+sq5*w_aux1);
    wdot_new2 = cq5*wdot2*(-3.673205103346573E-6)-cq5*wdot3-sq5*wdot1+qd5*(-cq5*w_aux1+sq5*w_aux2*3.673205103346573E-6+sq5*w_aux3);
    wdot_new3 = qdd5+wdot2-wdot3*3.673205103346573E-6;

    t2 = -linear_acc1;
    t3 = w_aux2*6.375E-3;
    t4 = wdot1*6.375E-3;
    t5 = wdot2*6.375E-3;
    t6 = w_aux1*w1*6.375E-3;
    t7 = w_aux1*w3*6.375E-3;
    t8 = w_aux3*2.0843E-1;
    t9 = wdot1*2.0843E-1;
    t10 = wdot3*2.0843E-1;
    t11 = w_aux1*w1*2.0843E-1;
    t12 = w_aux1*w2*2.0843E-1;
    t13 = -t8;
    t14 = -t9;
    t15 = -t10;
    t16 = t3+t13;
    t20 = t2+t5+t7+t12+t15;
    t17 = t16*w2;
    t18 = t16*w3;
    t19 = -t18;
    t21 = linear_acc3+t6+t14+t17;
    t22 = linear_acc2+t4+t11+t19;
    linear_acc_new1 = -cq5*t20-sq5*t21-sq5*t22*3.673205103346573E-6;
    linear_acc_new2 = -cq5*t21-cq5*t22*3.673205103346573E-6+sq5*t20;
    linear_acc_new3 = linear_acc2-linear_acc3*3.673205103346573E-6-t17*3.673205103346573E-6+t19+wdot1*6.375765606139691E-3+w_aux1*w1*2.084299765833175E-1;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[4][0]*w_aux2;
    t3 = com[4][0]*w_aux3;
    t4 = com[4][1]*w_aux1;
    t5 = com[4][1]*w_aux3;
    t6 = com[4][2]*w_aux1;
    t7 = com[4][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F5_1 = -mass_arr(4,0)*(-linear_acc1+com[4][1]*wdot3-com[4][2]*wdot2+t11*w2+t12*w3);
    PZsparse F5_2 = mass_arr(4,0)*(linear_acc2+com[4][0]*wdot3-com[4][2]*wdot1+t11*w1-t13*w3);
    PZsparse F5_3 = mass_arr(4,0)*(linear_acc3-com[4][0]*wdot2+com[4][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(4,0)(0,0)*w1;
    t3 = I_arr(4,0)(0,1)*w2;
    t4 = I_arr(4,0)(0,2)*w3;
    t5 = I_arr(4,0)(1,0)*w1;
    t6 = I_arr(4,0)(1,1)*w2;
    t7 = I_arr(4,0)(1,2)*w3;
    t8 = I_arr(4,0)(2,0)*w1;
    t9 = I_arr(4,0)(2,1)*w2;
    t10 = I_arr(4,0)(2,2)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N5_1 = I_arr(4,0)(0,0)*wdot1+I_arr(4,0)(0,1)*wdot2+I_arr(4,0)(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N5_2 = I_arr(4,0)(1,0)*wdot1+I_arr(4,0)(1,1)*wdot2+I_arr(4,0)(1,2)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N5_3 = I_arr(4,0)(2,0)*wdot1+I_arr(4,0)(2,1)*wdot2+I_arr(4,0)(2,2)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 6
    w_new1 = cq6*w1-sq6*w2*3.673205103346572E-6+sq6*w3;
    w_new2 = cq6*w2*(-3.673205103346572E-6)+cq6*w3-sq6*w1;
    w_new3 = qd6-w2-w3*3.673205103346572E-6;

    w_aux_new1 = cq6*w_aux1-sq6*w_aux2*3.673205103346572E-6+sq6*w_aux3;
    w_aux_new2 = cq6*w_aux2*(-3.673205103346572E-6)+cq6*w_aux3-sq6*w_aux1;
    w_aux_new3 = qda6-w_aux2-w_aux3*3.673205103346572E-6;

    wdot_new1 = cq6*wdot1-sq6*wdot2*3.673205103346572E-6+sq6*wdot3-qd6*(cq6*w_aux2*3.673205103346572E-6-cq6*w_aux3+sq6*w_aux1);
    wdot_new2 = cq6*wdot2*(-3.673205103346572E-6)+cq6*wdot3-sq6*wdot1-qd6*(cq6*w_aux1-sq6*w_aux2*3.673205103346572E-6+sq6*w_aux3);
    wdot_new3 = qdd6-wdot2-wdot3*3.673205103346572E-6;

    t2 = -linear_acc1;
    t3 = w_aux2*1.0593E-1;
    t4 = wdot1*1.0593E-1;
    t5 = wdot2*1.0593E-1;
    t6 = w_aux1*w1*1.0593E-1;
    t7 = w_aux1*w3*1.0593E-1;
    t8 = w_aux3*1.750499999999995E-4;
    t9 = wdot1*1.750499999999995E-4;
    t10 = wdot3*1.750499999999995E-4;
    t11 = w_aux1*w1*1.750499999999995E-4;
    t12 = w_aux1*w2*1.750499999999995E-4;
    t13 = -t11;
    t14 = -t12;
    t15 = t3+t8;
    t16 = t15*w2;
    t17 = t15*w3;
    t20 = t2+t5+t7+t10+t14;
    t18 = -t17;
    t19 = linear_acc3+t6+t9+t16;
    t21 = linear_acc2+t4+t13+t18;
    linear_acc_new1 = -cq6*t20+sq6*t19-sq6*t21*3.673205103346572E-6;
    linear_acc_new2 = cq6*t19-cq6*t21*3.673205103346572E-6+sq6*t20;
    linear_acc_new3 = -linear_acc2-linear_acc3*3.673205103346572E-6-t16*3.673205103346572E-6+t17-wdot1*1.059300006429946E-1+w_aux1*w1*1.74660897383402E-4;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[5][0]*w_aux2;
    t3 = com[5][0]*w_aux3;
    t4 = com[5][1]*w_aux1;
    t5 = com[5][1]*w_aux3;
    t6 = com[5][2]*w_aux1;
    t7 = com[5][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F6_1 = -mass_arr(5,0)*(-linear_acc1+com[5][1]*wdot3-com[5][2]*wdot2+t11*w2+t12*w3);
    PZsparse F6_2 = mass_arr(5,0)*(linear_acc2+com[5][0]*wdot3-com[5][2]*wdot1+t11*w1-t13*w3);
    PZsparse F6_3 = mass_arr(5,0)*(linear_acc3-com[5][0]*wdot2+com[5][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(5,0)(0,0)*w1;
    t3 = I_arr(5,0)(0,1)*w2;
    t4 = I_arr(5,0)(0,2)*w3;
    t5 = I_arr(5,0)(1,0)*w1;
    t6 = I_arr(5,0)(1,1)*w2;
    t7 = I_arr(5,0)(1,2)*w3;
    t8 = I_arr(5,0)(2,0)*w1;
    t9 = I_arr(5,0)(2,1)*w2;
    t10 = I_arr(5,0)(2,2)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N6_1 = I_arr(5,0)(0,0)*wdot1+I_arr(5,0)(0,1)*wdot2+I_arr(5,0)(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N6_2 = I_arr(5,0)(1,0)*wdot1+I_arr(5,0)(1,1)*wdot2+I_arr(5,0)(1,2)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N6_3 = I_arr(5,0)(2,0)*wdot1+I_arr(5,0)(2,1)*wdot2+I_arr(5,0)(2,2)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 7
    w_new1 = cq7*w1-sq7*w2*3.673205103346572E-6-sq7*w3;
    w_new2 = cq7*w2*(-3.673205103346572E-6)-cq7*w3-sq7*w1;
    w_new3 = qd7+w2-w3*3.673205103346572E-6;

    w_aux_new1 = cq7*w_aux1-sq7*w_aux2*3.673205103346572E-6-sq7*w_aux3;
    w_aux_new2 = cq7*w_aux2*(-3.673205103346572E-6)-cq7*w_aux3-sq7*w_aux1;
    w_aux_new3 = qda7+w_aux2-w_aux3*3.673205103346572E-6;

    wdot_new1 = cq7*wdot1-sq7*wdot2*3.673205103346572E-6-sq7*wdot3-qd7*(cq7*w_aux2*3.673205103346572E-6+cq7*w_aux3+sq7*w_aux1);
    wdot_new2 = cq7*wdot2*(-3.673205103346572E-6)-cq7*wdot3-sq7*wdot1+qd7*(-cq7*w_aux1+sq7*w_aux2*3.673205103346572E-6+sq7*w_aux3);
    wdot_new3 = qdd7+wdot2-wdot3*3.673205103346572E-6;

    t2 = -linear_acc1;
    t3 = w_aux3*1.0593E-1;
    t4 = wdot1*1.0593E-1;
    t5 = wdot3*1.0593E-1;
    t6 = w_aux1*w1*1.0593E-1;
    t7 = w_aux1*w2*1.0593E-1;
    t11 = w_aux2*1.750499999999995E-4;
    t12 = wdot1*1.750499999999995E-4;
    t13 = wdot2*1.750499999999995E-4;
    t14 = w_aux1*w1*1.750499999999995E-4;
    t15 = w_aux1*w3*1.750499999999995E-4;
    t8 = -t3;
    t9 = -t4;
    t10 = -t5;
    t17 = -w2*(t3-t11);
    t18 = -w3*(t3-t11);
    t19 = w3*(t3-t11);
    t16 = t8+t11;
    t20 = t2+t7+t10+t13+t15;
    t21 = linear_acc3+t9+t14+t17;
    t22 = linear_acc2+t6+t12+t19;
    linear_acc_new1 = -cq7*t20-sq7*t21-sq7*t22*3.673205103346572E-6;
    linear_acc_new2 = -cq7*t21-cq7*t22*3.673205103346572E-6+sq7*t20;
    linear_acc_new3 = linear_acc2-linear_acc3*3.673205103346572E-6+t19+wdot1*1.75439102616597E-4+w_aux1*w1*1.059299993570054E-1+w2*(t3-t11)*3.673205103346572E-6;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[6][0]*w_aux2;
    t3 = com[6][0]*w_aux3;
    t4 = com[6][1]*w_aux1;
    t5 = com[6][1]*w_aux3;
    t6 = com[6][2]*w_aux1;
    t7 = com[6][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F7_1 = -mass_arr(6,0)*(-linear_acc1+com[6][1]*wdot3-com[6][2]*wdot2+t11*w2+t12*w3);
    PZsparse F7_2 = mass_arr(6,0)*(linear_acc2+com[6][0]*wdot3-com[6][2]*wdot1+t11*w1-t13*w3);
    PZsparse F7_3 = mass_arr(6,0)*(linear_acc3-com[6][0]*wdot2+com[6][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(6,0)(0,0)*w1;
    t3 = I_arr(6,0)(0,1)*w2;
    t4 = I_arr(6,0)(0,2)*w3;
    t5 = I_arr(6,0)(1,0)*w1;
    t6 = I_arr(6,0)(1,1)*w2;
    t7 = I_arr(6,0)(1,2)*w3;
    t8 = I_arr(6,0)(2,0)*w1;
    t9 = I_arr(6,0)(2,1)*w2;
    t10 = I_arr(6,0)(2,2)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N7_1 = I_arr(6,0)(0,0)*wdot1+I_arr(6,0)(0,1)*wdot2+I_arr(6,0)(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N7_2 = I_arr(6,0)(1,0)*wdot1+I_arr(6,0)(1,1)*wdot2+I_arr(6,0)(1,2)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N7_3 = I_arr(6,0)(2,0)*wdot1+I_arr(6,0)(2,1)*wdot2+I_arr(6,0)(2,2)*wdot3-t11*w_aux2+t12*w_aux1;

    // joint 8
    w_new1 = w1;
    w_new2 = w2;
    w_new3 = w3;

    w_aux_new1 = w_aux1;
    w_aux_new2 = w_aux2;
    w_aux_new3 = w_aux3;

    wdot_new1 = wdot1;
    wdot_new2 = wdot2;
    wdot_new3 = wdot3;

    t2 = w_aux2/5.0E+2;
    t3 = w_aux3*(5.3E+1/5.0E+2);
    t4 = t2+t3;
    linear_acc_new1 = linear_acc1+wdot2/5.0E+2+wdot3*(5.3E+1/5.0E+2)-w_aux1*w2*(5.3E+1/5.0E+2)+(w_aux1*w3)/5.0E+2;
    linear_acc_new2 = linear_acc2-wdot1/5.0E+2+t4*w3+w_aux1*w1*(5.3E+1/5.0E+2);
    linear_acc_new3 = linear_acc3-wdot1*(5.3E+1/5.0E+2)-t4*w2-(w_aux1*w1)/5.0E+2;

    w1 = w_new1;
    w2 = w_new2;
    w3 = w_new3;
    w_aux1 = w_aux_new1;
    w_aux2 = w_aux_new2;
    w_aux3 = w_aux_new3;
    wdot1 = wdot_new1;
    wdot2 = wdot_new2;
    wdot3 = wdot_new3;
    linear_acc1 = linear_acc_new1;
    linear_acc2 = linear_acc_new2;
    linear_acc3 = linear_acc_new3;

    t2 = com[7][0]*w_aux2;
    t3 = com[7][0]*w_aux3;
    t4 = com[7][1]*w_aux1;
    t5 = com[7][1]*w_aux3;
    t6 = com[7][2]*w_aux1;
    t7 = com[7][2]*w_aux2;
    t8 = -t4;
    t9 = -t6;
    t10 = -t7;
    t11 = t2+t8;
    t12 = t3+t9;
    t13 = t5+t10;
    PZsparse F8_1 = -mass_arr(7,0)*(-linear_acc1+com[7][1]*wdot3-com[7][2]*wdot2+t11*w2+t12*w3);
    PZsparse F8_2 = mass_arr(7,0)*(linear_acc2+com[7][0]*wdot3-com[7][2]*wdot1+t11*w1-t13*w3);
    PZsparse F8_3 = mass_arr(7,0)*(linear_acc3-com[7][0]*wdot2+com[7][1]*wdot1+t12*w1+t13*w2);

    t2 = I_arr(7,0)(0,0)*w1;
    t3 = I_arr(7,0)(0,1)*w2;
    t4 = I_arr(7,0)(0,2)*w3;
    t5 = I_arr(7,0)(1,0)*w1;
    t6 = I_arr(7,0)(1,1)*w2;
    t7 = I_arr(7,0)(1,2)*w3;
    t8 = I_arr(7,0)(2,0)*w1;
    t9 = I_arr(7,0)(2,1)*w2;
    t10 = I_arr(7,0)(2,2)*w3;
    t11 = t2+t3+t4;
    t12 = t5+t6+t7;
    t13 = t8+t9+t10;
    PZsparse N8_1 = I_arr(7,0)(0,0)*wdot1+I_arr(7,0)(0,1)*wdot2+I_arr(7,0)(0,2)*wdot3-t12*w_aux3+t13*w_aux2;
    PZsparse N8_2 = I_arr(7,0)(1,0)*wdot1+I_arr(7,0)(1,1)*wdot2+I_arr(7,0)(1,2)*wdot3+t11*w_aux3-t13*w_aux1;
    PZsparse N8_3 = I_arr(7,0)(2,0)*wdot1+I_arr(7,0)(2,1)*wdot2+I_arr(7,0)(2,2)*wdot3-t11*w_aux2+t12*w_aux1;

    PZsparse f8_1 = F8_1;
    PZsparse f8_2 = F8_2;
    PZsparse f8_3 = F8_3;

    PZsparse n8_1 = N8_1+F8_3*com[7][1]-F8_2*com[7][2];
    PZsparse n8_2 = N8_2-F8_3*com[7][0]+F8_1*com[7][2];
    PZsparse n8_3 = N8_3+F8_2*com[7][0]-F8_1*com[7][1];

    PZsparse f7_1 = F7_1+f8_1;
    PZsparse f7_2 = F7_2+f8_2;
    PZsparse f7_3 = F7_3+f8_3;

    PZsparse n7_1 = N7_1-f8_2/5.0E+2-f8_3*(5.3E+1/5.0E+2)+n8_1+F7_3*com[6][1]-F7_2*com[6][2];
    PZsparse n7_2 = N7_2+f8_1/5.0E+2+n8_2-F7_3*com[6][0]+F7_1*com[6][2];
    PZsparse n7_3 = N7_3+f8_1*(5.3E+1/5.0E+2)+n8_3+F7_2*com[6][0]-F7_1*com[6][1];

    PZsparse f6_1 = F6_1+cq7*f7_1-f7_2*sq7;
    PZsparse f6_2 = F6_2+f7_3-cq7*f7_2*3.673205103346572E-6-f7_1*sq7*3.673205103346572E-6;
    PZsparse f6_3 = F6_3-f7_3*3.673205103346572E-6-cq7*f7_2-f7_1*sq7;

    PZsparse n6_1 = N6_1+f7_3*1.75439102616597E-4+F6_3*com[5][1]-F6_2*com[5][2]+cq7*f7_2*1.059299993570054E-1+cq7*n7_1+f7_1*sq7*1.059299993570054E-1-n7_2*sq7;
    PZsparse n6_2 = N6_2+n7_3-F6_3*com[5][0]+F6_1*com[5][2]-cq7*f7_1*1.750499999999995E-4-cq7*n7_2*3.673205103346572E-6+f7_2*sq7*1.750499999999995E-4-n7_1*sq7*3.673205103346572E-6;
    PZsparse n6_3 = N6_3-n7_3*3.673205103346572E-6+F6_2*com[5][0]-F6_1*com[5][1]+cq7*f7_1*1.0593E-1-cq7*n7_2-f7_2*sq7*1.0593E-1-n7_1*sq7;

    t2 = cq6*f6_2;
    t3 = f6_1*sq6;
    PZsparse f5_1 = F5_1+cq6*f6_1-f6_2*sq6;
    PZsparse f5_2 = F5_2-f6_3-t2*3.673205103346572E-6-t3*3.673205103346572E-6;
    PZsparse f5_3 = F5_3-f6_3*3.673205103346572E-6+t2+t3;

    t2 = cq6*n6_2;
    t3 = n6_1*sq6;
    PZsparse n5_1 = N5_1-f6_3*1.059300006429946E-1+F5_3*com[4][1]-F5_2*com[4][2]+cq6*f6_2*1.74660897383402E-4+cq6*n6_1+f6_1*sq6*1.74660897383402E-4-n6_2*sq6;
    PZsparse n5_2 = N5_2-n6_3-t2*3.673205103346572E-6-t3*3.673205103346572E-6-F5_3*com[4][0]+F5_1*com[4][2]-cq6*f6_1*1.0593E-1+f6_2*sq6*1.0593E-1;
    PZsparse n5_3 = N5_3-n6_3*3.673205103346572E-6+t2+t3+F5_2*com[4][0]-F5_1*com[4][1]-cq6*f6_1*1.750499999999995E-4+f6_2*sq6*1.750499999999995E-4;

    PZsparse f4_1 = F4_1+cq5*f5_1-f5_2*sq5;
    PZsparse f4_2 = F4_2+f5_3-cq5*f5_2*3.673205103346573E-6-f5_1*sq5*3.673205103346573E-6;
    PZsparse f4_3 = F4_3-f5_3*3.673205103346573E-6-cq5*f5_2-f5_1*sq5;

    PZsparse n4_1 = N4_1+f5_3*6.375765606139691E-3+F4_3*com[3][1]-F4_2*com[3][2]+cq5*f5_2*2.084299765833175E-1+cq5*n5_1+f5_1*sq5*2.084299765833175E-1-n5_2*sq5;
    PZsparse n4_2 = N4_2+n5_3-F4_3*com[3][0]+F4_1*com[3][2]-cq5*f5_1*6.375E-3-cq5*n5_2*3.673205103346573E-6+f5_2*sq5*6.375E-3-n5_1*sq5*3.673205103346573E-6;
    PZsparse n4_3 = N4_3-n5_3*3.673205103346573E-6+F4_2*com[3][0]-F4_1*com[3][1]+cq5*f5_1*2.0843E-1-cq5*n5_2-f5_2*sq5*2.0843E-1-n5_1*sq5;

    t2 = cq4*f4_2;
    t3 = f4_1*sq4;
    PZsparse f3_1 = F3_1+cq4*f4_1-f4_2*sq4;
    PZsparse f3_2 = F3_2-f4_3-t2*3.673205103346573E-6-t3*3.673205103346573E-6;
    PZsparse f3_3 = F3_3-f4_3*3.673205103346573E-6+t2+t3;

    t2 = cq4*n4_2;
    t3 = n4_1*sq4;
    PZsparse n3_1 = N3_1-f4_3*2.103800234166825E-1+F3_3*com[2][1]-F3_2*com[2][2]+cq4*f4_2*6.374227231110358E-3+cq4*n4_1+f4_1*sq4*6.374227231110358E-3-n4_2*sq4;
    PZsparse n3_2 = N3_2-n4_3-t2*3.673205103346573E-6-t3*3.673205103346573E-6-F3_3*com[2][0]+F3_1*com[2][2]-cq4*f4_1*2.1038E-1+f4_2*sq4*2.1038E-1;
    PZsparse n3_3 = N3_3-n4_3*3.673205103346573E-6+t2+t3+F3_2*com[2][0]-F3_1*com[2][1]-cq4*f4_1*6.375E-3+f4_2*sq4*6.375E-3;

    PZsparse f2_1 = F2_1+cq3*f3_1-f3_2*sq3;
    PZsparse f2_2 = F2_2+f3_3-cq3*f3_2*3.673205103346574E-6-f3_1*sq3*3.673205103346574E-6;
    PZsparse f2_3 = F2_3-f3_3*3.673205103346574E-6-cq3*f3_2-f3_1*sq3;

    PZsparse n2_1 = N2_1+f3_3*6.375772768889642E-3+F2_3*com[1][1]-F2_2*com[1][2]+cq3*f3_2*2.103799765833175E-1+cq3*n3_1+f3_1*sq3*2.103799765833175E-1-n3_2*sq3;
    PZsparse n2_2 = N2_2+n3_3-F2_3*com[1][0]+F2_1*com[1][2]-cq3*f3_1*6.375E-3-cq3*n3_2*3.673205103346574E-6+f3_2*sq3*6.375E-3-n3_1*sq3*3.673205103346574E-6;
    PZsparse n2_3 = N2_3-n3_3*3.673205103346574E-6+F2_2*com[1][0]-F2_1*com[1][1]+cq3*f3_1*2.1038E-1-cq3*n3_2-f3_2*sq3*2.1038E-1-n3_1*sq3;

    // t2 = cq2*f2_2;
    // t3 = f2_1*sq2;
    // PZsparse f1_1 = F1_1+cq2*f2_1-f2_2*sq2;
    // PZsparse f1_2 = F1_2-f2_3-t2*3.673205103346574E-6-t3*3.673205103346574E-6;
    // PZsparse f1_3 = F1_3-f2_3*3.673205103346574E-6+t2+t3;

    t2 = cq2*n2_2;
    t3 = n2_1*sq2;
    // PZsparse n1_1 = N1_1-f2_3*1.283800197434774E-1+F1_3*com[0][1]-F1_2*com[0][2]+cq2*f2_2*5.374528433928832E-3+cq2*n2_1+f2_1*sq2*5.374528433928832E-3-n2_2*sq2;
    // PZsparse n1_2 = N1_2-n2_3-t2*3.673205103346574E-6-t3*3.673205103346574E-6-F1_3*com[0][0]+F1_1*com[0][2]-cq2*f2_1*1.2838E-1+f2_2*sq2*1.2838E-1;
    PZsparse n1_3 = N1_3-n2_3*3.673205103346574E-6+t2+t3+F1_2*com[0][0]-F1_1*com[0][1]-cq2*f2_1*5.375E-3+f2_2*sq2*5.375E-3;

    u(0,s_ind) = n1_3 + damping[0] * traj->qd_des(0, s_ind) + armature[0] * traj->qdda_des(0, s_ind);
    u(1,s_ind) = n2_3 + damping[1] * traj->qd_des(1, s_ind) + armature[1] * traj->qdda_des(1, s_ind);
    u(2,s_ind) = n3_3 + damping[2] * traj->qd_des(2, s_ind) + armature[2] * traj->qdda_des(2, s_ind);
    u(3,s_ind) = n4_3 + damping[3] * traj->qd_des(3, s_ind) + armature[3] * traj->qdda_des(3, s_ind);
    u(4,s_ind) = n5_3 + damping[4] * traj->qd_des(4, s_ind) + armature[4] * traj->qdda_des(4, s_ind);
    u(5,s_ind) = n6_3 + damping[5] * traj->qd_des(5, s_ind) + armature[5] * traj->qdda_des(5, s_ind);
    u(6,s_ind) = n7_3 + damping[6] * traj->qd_des(6, s_ind) + armature[6] * traj->qdda_des(6, s_ind);

    PZsparseArray f8(3, 1);
    f8(0,0) = f8_1;
    f8(1,0) = f8_2;
    f8(2,0) = f8_3;
    f_c(0,s_ind) = stack(f8); // not sure how to assign these
    PZsparseArray n8(3, 1);
    n8(0,0) = n8_1;
    n8(1,0) = n8_2;
    n8(2,0) = n8_3;
    n_c(0,s_ind) = stack(n8); // not sure how to assign these
}

#endif